#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/logistic_posterior_prob_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidForward(const int n, const Dtype* in_first, 
    const Dtype* in_second, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype temp_val = in_first[index] - in_second[index]
    out[index] = 1. / (1. + exp(-temp_val));
  }
}

template <typename Dtype>
void LogisticPosteriorProbLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  if (bottom.size() != 2) { return; }
  const Dtype* bottom_data_first = bottom[0]->gpu_data();
  const Dtype* bottom_data_second = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data_first, bottom_data_second, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SigmoidBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index];
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x);
  }
}

template <typename Dtype>
void LogisticPosteriorProbLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (bottom.size() != 2) { return; }
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  for (int i = 0; i < bottom.size(); ++i) {
    if (propagate_down[i]) {
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      const int count = bottom[i]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      SigmoidBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, top_data, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(LogisticPosteriorProbLayer);


}  // namespace caffe
